
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include "Buffer.cuh"
#include "NeuralNetwork.cuh"


template <class floot=float>
void test_nn()
{
	cudann::NeuralNetwork<floot> nn({ 16, 8, 4, 3 });
	nn.init_host();
	nn.fill_random_weights();
	nn.init_device();
	nn.print_info();
	nn.print_weights();


	cudann::Buffer<floot> input(16);
	input.malloc_host();
	input.fill_host({0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15});

	const cudann::Buffer<floot> & res = nn.predict_host(input);

	utils::print_collection(std::cout, res.host_compact());
}

int main()
{
	test_nn();
}