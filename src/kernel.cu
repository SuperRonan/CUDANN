
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include "Buffer.cuh"
#include "NeuralNetwork.cuh"


int main()
{
	cudann::NeuralNetwork<float> nn({ 16, 8, 4, 3 });
	nn.init_host();
	nn.fill_random_weights();
	nn.init_device();
	nn.print_info();
	nn.print_weights();
}