
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include "Buffer.cuh"
#include "NeuralNetwork.cuh"





template <class floot = float>
void test_xor()
{
	cudann::NeuralNetwork<floot> xornn({ 2, 2, 1 });
	xornn.init_host();
	xornn.fill_random_weights();
	xornn.print_info();
	xornn.print_weights();

	std::vector<std::pair<cudann::Buffer<floot>, cudann::Buffer<floot>>> training_set;
	training_set.reserve(4);
	for (char x = 0; x < 2; ++x)			


	{
		for (char y = 0; y < 2; ++y)
		{

			training_set.emplace_back(2, 1);
			cudann::Buffer<floot> & example = training_set.back().first;
			cudann::Buffer<floot> & truth = training_set.back().second;
			
			example.malloc_host();
			truth.malloc_host();

			example.fill_host({ floot(x), floot(y) });
			truth.fill_host({ floot(x ^ y) });
		}
	}

	xornn.fit_host(training_set, 100);

	xornn.print_info();
	xornn.print_weights();

}

template <class floot=float>
void test_nn()
{
	const unsigned int input_size = 3;
	cudann::NeuralNetwork<floot> nn({ input_size, 2, 1 });
	nn.init_host();
	nn.fill_random_weights();
	nn.init_device();
	nn.print_info();
	nn.print_weights();


	cudann::Buffer<floot> input(input_size);
	input.malloc_host();
	input.fill_host({-1, 1, 2});

	const cudann::Buffer<floot> & res = nn.predict_host(input);

	utils::print_collection(std::cout, res.host_compact());
}

int main()
{
	test_nn();
	test_xor();
}